#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__
void flash_attention_2_forward_kernel(
    const float* Q,
    const float* K,
    const float* V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float* L,
    float* O
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    // Offset into Q,K,V,O - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for L

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d;  // size of Qi, Kj, Vj
    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size * 2];
    float* S = &sram[tile_size * 3];

    for (int i = 0; i < Tr; i++) { // Q is outer loop

        // Load Qi from HBM to SRAM
        for (int x = 0; x < d; x++) {
            Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
        }
        // Initialize l and m
        float row_m_prev = -INFINITY;
        float row_l_prev = 0;

        for (int j = 0; j < Tc; j++) { // K, V is inner loop
            // Load Kj, Vj from HBM to SRAM
            for (int x = 0; x < d; x++) {
                Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
                Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
            }
            __syncthreads();

            // compute S = QK^T
            // S_i^j = softmax_scale * QiKj^T
            // S_i^j[tx][y] = softmax_scale * Sum_{x = 0}^{d-1} Qi[tx][x] * Kj[y][x]
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++) {
                float sum = 0;
                for (int x = 0; x < d; x++)
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                sum *= softmax_scale;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // compute local max
            // m_i^j = max(m_i^j-1, row_max(S_i^j))
            float row_m_new = max(row_m_prev, row_m);


            // compute local sum
            // P_i^j = exp(S_i^j - m_i^j)
            // P_i^j[tx][y] = exp(S_i^j[tx][y] - m_i^j)
            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m_new);
                row_l += S[(Bc * tx) + y];
            }

            // l_i^j = (exp(m_i^j-1 - m_i^j) * l_i^j-1) + row_sum(P_i^j)
            float row_m_exp = __expf(row_m_prev - row_m_new);
            float row_l_new = (row_m_exp * row_l_prev) + row_l;

            // compute P@V and O
            // O_i^j = diag(exp(m_i^j-1 - m_i^j))^-1 * O_i^j-1 + P_i^jVj
            for (int x = 0; x < d; x++) {
                float pv = 0;  // Pij * Vj
                for (int y = 0; y < Bc; y++) {
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = \
                    row_m_exp * O[qkv_offset + (tile_size * i) + (tx * d) + x] + pv;
            }

            // Update m and l
            row_m_prev = row_m_new;
            row_l_prev = row_l_new;
        } // end K, V loop

        // O_i = diag(l_i^{Tc})^-1 * O_i^{Tc}
        for (int x = 0; x < d; x++)
            O[qkv_offset + (tile_size * i) + (tx * d) + x] /= row_l_prev;
        // L_i = m_i^{Tc} + log(l_i^{Tc})
        L[lm_offset + (Br * i) + tx] = row_m_prev + __logf(row_l_prev);
    }
}

std::vector<torch::Tensor> flash_attention_2_forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    // TODO: determine Bc, Br dynamically
    const int Bc = 32; const int Br = 32;

    const int B = Q.size(0); const int nh = Q.size(1);
    const int N = Q.size(2); const int d = Q.size(3);

    const int Tc = ceil((float) N / Bc); const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);

    // Initialize O, L to HBM
    auto O = torch::zeros_like(Q);
    auto L = torch::zeros({B, nh, N});
    torch::Device device(torch::kCUDA);
    L = L.to(device);

    // Calculate SRAM size needed per block
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi
    const int sram_size =
        (2 * col_tile_size * sizeof(float))  // SRAM size for Kj, Vj
        + (row_tile_size * sizeof(float))  // SRAM size for Qi
        + (Bc * Br * sizeof(float));  // SRAM size for S
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(Br);  // Br threads per block

    flash_attention_2_forward_kernel<<<grid_dim, block_dim, sram_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        L.data_ptr<float>(), O.data_ptr<float>()
    );
    return {O, L};
}
~

